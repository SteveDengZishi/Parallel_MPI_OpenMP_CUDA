#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void getmaxcu(unsigned int* numbers_device, unsigned int* max_device, unsigned int size){

    __device__ __shared__ unsigned int shared_num[1024];

    //copy from device global memory to device shared memory
    shared_num[threadIdx.x] = numbers_device[blockDim.x * blockIdx.x + threadIdx.x];
    __syncthreads();

    //use reduction to find max
    unsigned int tid=threadIdx.x;
    unsigned int i;
    for(i=blockDim.x>>1;i>0;i>>=1){
      __syncthreads();
      if(tid<i){
        shared_num[tid]=max(shared_num[tid],shared_num[tid+i]);
      }
    }
    __syncthreads();
    //shared_num[0] is the maximum by now in each blocks
    if(threadIdx.x==0){
      atomicMax(max_device, shared_num[0]);
    }
}

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);
    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));

    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++) numbers[i] = rand()  % size;  

    /*
    //checking and printing device properties
    int device;
    hipDeviceProp_t cuda_properties;
    hipGetDevice(&device);
    hipGetDeviceProperties(&cuda_properties,device);
    printf("Device Properties for %s\n",cuda_properties.name);
    printf("================================================\n");
    printf("Total Global Memory Size is %u\n", cuda_properties.totalGlobalMem);
    printf("Shared Memory per block is %u\n", cuda_properties.sharedMemPerBlock);
    printf("Warp Size is %d and register per block is %d\n", cuda_properties.warpSize, cuda_properties.regsPerBlock);
    printf("Max threads per block is %d\n", cuda_properties.maxThreadsPerBlock);
    printf("================================================\n");
    */
    
    //allocating on the device
    unsigned int max=0;
    unsigned int * numbers_device;
    unsigned int * max_device;
    hipError_t error = hipMalloc((void**)&numbers_device, size * sizeof(unsigned int));

    //error handling
    if(error != hipSuccess){ // print the CUDA error message and exit printf("CUDA error: %s\n",
      hipGetErrorString(error);
      exit(-1);
    }

    error = hipMalloc((void**)&max_device, sizeof(unsigned int));

    if(error != hipSuccess){ // print the CUDA error message and exit printf("CUDA error: %s\n",
      hipGetErrorString(error);
      exit(-1);
    }

    hipMemcpy(numbers_device, numbers, size * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(max_device, &max, sizeof(unsigned int), hipMemcpyHostToDevice);

    //lauch pre-defined kernel code
    int block_size=1024;
    int block_num=ceil((double)size/(double)block_size);

    //invoke kernel
    getmaxcu<<<block_num,block_size>>>(numbers_device,max_device,size);
    hipDeviceSynchronize();
    //copy max_device back to host
    hipMemcpy(&max, max_device, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("The maximum number in the array is: %u\n", max);

    //memory management
    free(numbers);
    hipFree(numbers_device);
    hipFree(max_device);
    exit(0); 
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array

unsigned int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}
*/
